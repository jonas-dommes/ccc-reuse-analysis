#include "calls_access_pattern.cuh"
#include "utility.cuh"
#include "test_access_pattern.cuh"

#include <stdio.h>
#include <assert.h>
#include <time.h>

#define BLOCKDIM_X 64
#define BLOCKDIM_Y 16
#define BLOCKDIM_Z 1
#define GRIDDIM_X 512
#define GRIDDIM_Y 512
#define GRIDDIM_Z 1


int call_D2_ap_transpose_block() {

	int datasize = BLOCKDIM_X * BLOCKDIM_Y;

	// Prepare Kernel dimensions
	dim3 dimBlock(BLOCKDIM_X, BLOCKDIM_Y, BLOCKDIM_Z);
	dim3 dimGrid(GRIDDIM_X, GRIDDIM_Y, GRIDDIM_Z);

	struct data_float data;
	init_data_float(&data, datasize);

	// Events for timing
	hipEvent_t startEvent, stopEvent;
	checkCuda(hipEventCreate(&startEvent));
	checkCuda(hipEventCreate(&stopEvent));
	float ms;

	// Run Kernel
	checkCuda(hipEventRecord(startEvent, 0));
	D2_ap_transpose_block<<<dimGrid, dimBlock>>>(data.d_idata, data.d_odata);
	checkCuda(hipGetLastError());
	checkCuda(hipEventRecord(stopEvent, 0));
	checkCuda(hipEventSynchronize(stopEvent));
	checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
	checkCuda(hipMemcpy(data.h_odata, data.d_odata, datasize * sizeof(float), hipMemcpyDeviceToHost));

	// Cleanup
	checkCuda(hipEventDestroy(startEvent));
	checkCuda(hipEventDestroy(stopEvent));
	free_data_float(&data);

	return (int) ms;
}

int call_D2_ap_transpose() {

	int datasize = BLOCKDIM_X * BLOCKDIM_Y * BLOCKDIM_Z * GRIDDIM_X * GRIDDIM_Y * GRIDDIM_Z;

	// Prepare Kernel dimensions
	dim3 dimBlock(BLOCKDIM_X, BLOCKDIM_Y, BLOCKDIM_Z);
	dim3 dimGrid(GRIDDIM_X, GRIDDIM_Y, GRIDDIM_Z);

	struct data_float data;
	init_data_float(&data, datasize);

	// Events for timing
	hipEvent_t startEvent, stopEvent;
	checkCuda(hipEventCreate(&startEvent));
	checkCuda(hipEventCreate(&stopEvent));
	float ms;

	// Run Kernel
	checkCuda(hipEventRecord(startEvent, 0));
	D2_ap_transpose<<<dimGrid, dimBlock>>>(data.d_idata, data.d_odata);
	checkCuda(hipGetLastError());
	checkCuda(hipEventRecord(stopEvent, 0));
	checkCuda(hipEventSynchronize(stopEvent));
	checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
	checkCuda(hipMemcpy(data.h_odata, data.d_odata, datasize * sizeof(float), hipMemcpyDeviceToHost));

	// Cleanup
	checkCuda(hipEventDestroy(startEvent));
	checkCuda(hipEventDestroy(stopEvent));
	free_data_float(&data);

	return (int) ms;
}

int call_D2_ap_stepsize() {

	int datasize = BLOCKDIM_X * BLOCKDIM_Y * BLOCKDIM_Z * GRIDDIM_X * GRIDDIM_Y * GRIDDIM_Z;

	// Prepare Kernel dimensions
	dim3 dimBlock(BLOCKDIM_X, BLOCKDIM_Y, BLOCKDIM_Z);
	dim3 dimGrid(GRIDDIM_X, GRIDDIM_Y, GRIDDIM_Z);

	struct data_float data;
	init_data_float(&data, datasize);

	// Events for timing
	hipEvent_t startEvent, stopEvent;
	checkCuda(hipEventCreate(&startEvent));
	checkCuda(hipEventCreate(&stopEvent));
	float ms;

	// Run Kernel
	checkCuda(hipEventRecord(startEvent, 0));
	D2_ap_stepsize<<<dimGrid, dimBlock>>>(data.d_idata, data.d_odata);
	checkCuda(hipGetLastError());
	checkCuda(hipEventRecord(stopEvent, 0));
	checkCuda(hipEventSynchronize(stopEvent));
	checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
	checkCuda(hipMemcpy(data.h_odata, data.d_odata, datasize * sizeof(float), hipMemcpyDeviceToHost));

	// Cleanup
	checkCuda(hipEventDestroy(startEvent));
	checkCuda(hipEventDestroy(stopEvent));
	free_data_float(&data);

	return (int) ms;
}
