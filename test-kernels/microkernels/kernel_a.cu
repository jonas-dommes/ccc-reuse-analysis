#include "hip/hip_runtime.h"
/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*	notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*	notice, this list of conditions and the following disclaimer in the
*	documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*	contributors may be used to endorse or promote products derived
*	from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <stdio.h>
#include <assert.h>
#include <time.h>

#include "utility.h"

#define DEBUG 1

// Data reuse of first few entries
__global__ void kernel_a(float *odata, const float *idata, int work_per_thread) {

	int offset = blockIdx.x * blockDim.x * work_per_thread + threadIdx.x;

	for (int i = 0; i < work_per_thread; i++) {
		int index = i * blockDim.x + offset;
		odata[index] = idata[index] * idata[index] - 1;
	}

	if (offset < 32) {
		odata[offset] = -odata[offset];
	}
}

// Using data multiple times
__global__ void kernel_b(float *odata, const float *idata, int work_per_thread) {

	int offset = blockIdx.x * blockDim.x * work_per_thread + threadIdx.x;

	for (int i = 0; i < work_per_thread; i++) {
		int index = i * blockDim.x + offset;
		odata[index] = idata[index] * idata[index] - 1;
		odata[index] += odata[index] * idata[index] - 1;
	}

	if (offset < 32) {
		odata[offset] = -odata[offset];
	}
}

// Use shared memory


int main(int argc, char **argv) {

	// Handle and print arguments
	print_args(argc, argv);
	int num_blocks = atoi(argv[1]);
	int threads_per_block = atoi(argv[2]);
	int work_per_thread = atoi(argv[3]);

	int data_points = num_blocks * threads_per_block * work_per_thread;
	printf("data_points = %d\n", data_points);
	printf("num_threads = %d\n", num_blocks * threads_per_block);

	// Prepare Kernel dimensions
	dim3 dimGrid(num_blocks, 1, 1);
	dim3 dimBlock(threads_per_block, 1, 1);

	// Prepare host data structures
	float *h_idata = (float*) calloc(data_points, sizeof(float));
	float *h_odata = (float*) calloc(data_points, sizeof(float));

	// Initiallize input array
	init_random(h_idata, data_points);

	// // Calculate reference
	// float *reference = (float*) calloc(data_points, sizeof(float));
	//
	// clock_t begin = clock();
	//
	// for (int i = 0; i < data_points; i++) {
	// 	reference[i] = h_idata[i] * h_idata[i] - 1;
	// 	reference[i] += reference[i] * h_idata[i] - 1;
	// }
	//
	// for (int i = 0; i < 32; i++) {
	// 	reference[i] = -reference[i];
	//
	// clock_t end = clock();
	// double time_spent = (double)(end - begin) / CLOCKS_PER_SEC * 1000;
	// printf("Calculated reference in %.5f ms\n", time_spent);

	// Prepare device data structures
	float *d_idata, *d_odata;
	checkCuda(hipMalloc(&d_idata, data_points * sizeof(float)));
	checkCuda(hipMalloc(&d_odata, data_points * sizeof(float)));
	checkCuda(hipMemcpy(d_idata, h_idata, data_points * sizeof(float), hipMemcpyHostToDevice));
	checkCuda(hipMemset(d_odata, 0, data_points * sizeof(float)));

	// // Events for timing
	// hipEvent_t startEvent, stopEvent;
	// checkCuda(hipEventCreate(&startEvent));
	// checkCuda(hipEventCreate(&stopEvent));
	// float ms;

	// Run Kernel a
	// checkCuda(hipEventRecord(startEvent, 0));
	kernel_a<<<dimGrid, dimBlock>>>(d_odata, d_idata, work_per_thread);
	checkCuda(hipGetLastError());
	// checkCuda(hipEventRecord(stopEvent, 0));
	// checkCuda(hipEventSynchronize(stopEvent));
	// checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
	checkCuda(hipMemcpy(h_odata, d_odata, data_points * sizeof(float), hipMemcpyDeviceToHost));

	// Run Kernel b
	// checkCuda(hipEventRecord(startEvent, 0));
	kernel_b<<<dimGrid, dimBlock>>>(d_odata, d_idata, work_per_thread);
	checkCuda(hipGetLastError());
	// checkCuda(hipEventRecord(stopEvent, 0));
	// checkCuda(hipEventSynchronize(stopEvent));
	// checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
	checkCuda(hipMemcpy(h_odata, d_odata, data_points * sizeof(float), hipMemcpyDeviceToHost));


	// Analyse
	// int is_correct = check_result(reference, h_odata, data_points);
	// if (is_correct != -1) {
	// 	printf("Wrong result:   h_idata[%d] = %.20f\n\n",is_correct, h_idata[is_correct]);
	// } else {
	// 	printf("Correct result after %.5f ms\n", ms);
	// }

	// Cleanup
	// checkCuda(hipEventDestroy(startEvent));
	// checkCuda(hipEventDestroy(stopEvent));
	checkCuda(hipFree(d_idata));
	checkCuda(hipFree(d_odata));
	free(h_idata);
	free(h_odata);
	// free(reference);
}
