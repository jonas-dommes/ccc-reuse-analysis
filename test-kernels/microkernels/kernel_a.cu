/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*	notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*	notice, this list of conditions and the following disclaimer in the
*	documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*	contributors may be used to endorse or promote products derived
*	from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <time.h>

#define DEBUG 1
#define eps 10e-6

__global__ void kernel_a(float *odata, const float *idata, int work_per_thread) {

	int offset = blockIdx.x * blockDim.x * work_per_thread + threadIdx.x;

	for (int i = 0; i < work_per_thread; i++) {
		int index = i * blockDim.x + offset;
		odata[index] = idata[index] * idata[index] - 1;
	}

	if (offset < 32) {
		odata[offset] = -odata[offset]; // TODO: *(-1)?
		// Alternative: Reduce to one datapoint?
	}
}


// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline hipError_t checkCuda(hipError_t result) {

	#if defined(DEBUG) || defined(_DEBUG)
	if (result != hipSuccess) {
		fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
		assert(result == hipSuccess);
	}
	#endif
	return result;
}

// Check result for errors, return 1 if result differs
int check_result(const float *reference, const float *result, int n) {

	for (int i = 0; i < n; i++) {
		if (abs(reference[i] - result[i]) > eps) {
			printf("Wrong result: reference[%d] = %.20f\n", i, reference[i]);
			printf("Wrong result:    result[%d] = %.20f\n",i, result[i]);
			return i;
		}
	}
	return -1;
}

void print_args(int argc, char **argv) {

	if (argc != 4) {
		printf("Error: Format should be: ./copy num_blocks threads_per_block work_per_thread \n");
		exit(1);
	} else {
		printf("num_blocks        = %d\n", atoi(argv[1]));
		printf("threads_per_block = %d\n", atoi(argv[2]));
		printf("work_per_thread   = %d\n", atoi(argv[3]));
	}
}

// Initiallize array with random float between 0 and 10
void init_random(float *array, int n) {

	srand(42);

	for (int i = 0; i < n; i++) {
		array[i] = ((float) rand()/(float) (RAND_MAX)) * 10;
	}
}

int main(int argc, char **argv) {

	// Handle and print arguments
	print_args(argc, argv);
	int num_blocks = atoi(argv[1]);
	int threads_per_block = atoi(argv[2]);
	int work_per_thread = atoi(argv[3]);

	int data_points = num_blocks * threads_per_block * work_per_thread;
	printf("data_points = %d\n", data_points);
	printf("num_threads = %d\n", num_blocks * threads_per_block);

	// Prepare Kernel dimensions
	dim3 dimGrid(num_blocks, 1, 1);
	dim3 dimBlock(threads_per_block, 1, 1);

	// Prepare host data structures
	float *h_idata = (float*) calloc(data_points, sizeof(float));
	float *h_odata = (float*) calloc(data_points, sizeof(float));
	float *reference = (float*) calloc(data_points, sizeof(float));

	// Initiallize input array
	init_random(h_idata, data_points);

	// Calculate reference
	clock_t begin = clock();

	for (int i = 0; i < data_points; i++) {
		reference[i] = h_idata[i] * h_idata[i] - 1;
	}

	for (int i = 0; i < 32; i++) {
		reference[i] = -reference[i]; // TODO: *(-1)?
		// Alternative: Reduce to one datapoint?
	}

	clock_t end = clock();
	double time_spent = (double)(end - begin) / CLOCKS_PER_SEC * 1000;
	printf("Calculated reference in %.5f ms\n", time_spent);

	// Prepare device data structures
	float *d_idata, *d_odata;
	checkCuda(hipMalloc(&d_idata, data_points * sizeof(float)));
	checkCuda(hipMalloc(&d_odata, data_points * sizeof(float)));
	checkCuda(hipMemcpy(d_idata, h_idata, data_points * sizeof(float), hipMemcpyHostToDevice));
	checkCuda(hipMemset(d_odata, 0, data_points * sizeof(float)));

	// Events for timing
	hipEvent_t startEvent, stopEvent;
	checkCuda(hipEventCreate(&startEvent));
	checkCuda(hipEventCreate(&stopEvent));
	float ms;

	// Run Kernel
	checkCuda(hipEventRecord(startEvent, 0));
	kernel_a<<<dimGrid, dimBlock>>>(d_odata, d_idata, work_per_thread);
	checkCuda(hipGetLastError());
	checkCuda(hipEventRecord(stopEvent, 0));
	checkCuda(hipEventSynchronize(stopEvent));
	checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
	checkCuda(hipMemcpy(h_odata, d_odata, data_points * sizeof(float), hipMemcpyDeviceToHost));

	// Analyse
	int is_correct = check_result(reference, h_odata, data_points);
	if (is_correct != -1) {
		printf("Wrong result:    h_idata[%d] = %.20f\n\n",is_correct, h_idata[is_correct]);
	} else {
		printf("Correct result after %.5f ms\n", ms);
	}

	// Cleanup
	checkCuda(hipEventDestroy(startEvent));
	checkCuda(hipEventDestroy(stopEvent));
	checkCuda(hipFree(d_idata));
	checkCuda(hipFree(d_odata));
	free(h_idata);
	free(h_odata);
	free(reference);
}
