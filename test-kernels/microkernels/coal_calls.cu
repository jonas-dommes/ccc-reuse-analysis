#include <stdio.h>
#include <assert.h>
#include <time.h>

#include "utility.cuh"

#include "test_coalescing.cuh"

#define BLOCKDIM_X 512
#define BLOCKDIM_Y 1
#define BLOCKDIM_Z 1
#define GRIDDIM_X 65535
#define GRIDDIM_Y 1
#define GRIDDIM_Z 1


int run_D1_copy_coal100() {

	int datasize = BLOCKDIM_X * GRIDDIM_X;

	// Prepare Kernel dimensions
	dim3 dimGrid(GRIDDIM_X, 1, 1);
	dim3 dimBlock(BLOCKDIM_X, 1, 1);

	struct call_data data;
	init_call_data(&data, datasize);

	// Events for timing
	hipEvent_t startEvent, stopEvent;
	checkCuda(hipEventCreate(&startEvent));
	checkCuda(hipEventCreate(&stopEvent));
	float ms;

	// Run Kernel
	checkCuda(hipEventRecord(startEvent, 0));
	D1_copy_coal100<<<dimGrid, dimBlock>>>(data.d_odata, data.d_idata);
	checkCuda(hipGetLastError());
	checkCuda(hipEventRecord(stopEvent, 0));
	checkCuda(hipEventSynchronize(stopEvent));
	checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
	checkCuda(hipMemcpy(data.h_odata, data.d_odata, datasize * sizeof(float), hipMemcpyDeviceToHost));

	// Cleanup
	checkCuda(hipEventDestroy(startEvent));
	checkCuda(hipEventDestroy(stopEvent));
	free_call_data(&data);

	return (int) ms;
}

int run_D1_copy_coal50() {

	int datasize = BLOCKDIM_X * GRIDDIM_X * 2;

	// Prepare Kernel dimensions
	dim3 dimGrid(GRIDDIM_X, 1, 1);
	dim3 dimBlock(BLOCKDIM_X, 1, 1);

	struct call_data data;
	init_call_data(&data, datasize);

	// Events for timing
	hipEvent_t startEvent, stopEvent;
	checkCuda(hipEventCreate(&startEvent));
	checkCuda(hipEventCreate(&stopEvent));
	float ms;

	// Run Kernel
	checkCuda(hipEventRecord(startEvent, 0));
	D1_copy_coal50<<<dimGrid, dimBlock>>>(data.d_odata, data.d_idata);
	checkCuda(hipGetLastError());
	checkCuda(hipEventRecord(stopEvent, 0));
	checkCuda(hipEventSynchronize(stopEvent));
	checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
	checkCuda(hipMemcpy(data.h_odata, data.d_odata, datasize * sizeof(float), hipMemcpyDeviceToHost));

	// Cleanup
	checkCuda(hipEventDestroy(startEvent));
	checkCuda(hipEventDestroy(stopEvent));
	free_call_data(&data);

	return (int) ms;
}

int run_D1_copy_coal25() {

	int datasize = BLOCKDIM_X * GRIDDIM_X * 4;

	// Prepare Kernel dimensions
	dim3 dimGrid(GRIDDIM_X, 1, 1);
	dim3 dimBlock(BLOCKDIM_X, 1, 1);

	struct call_data data;
	init_call_data(&data, datasize);

	// Events for timing
	hipEvent_t startEvent, stopEvent;
	checkCuda(hipEventCreate(&startEvent));
	checkCuda(hipEventCreate(&stopEvent));
	float ms;

	// Run Kernel
	checkCuda(hipEventRecord(startEvent, 0));
	D1_copy_coal25<<<dimGrid, dimBlock>>>(data.d_odata, data.d_idata);
	checkCuda(hipGetLastError());
	checkCuda(hipEventRecord(stopEvent, 0));
	checkCuda(hipEventSynchronize(stopEvent));
	checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
	checkCuda(hipMemcpy(data.h_odata, data.d_odata, datasize * sizeof(float), hipMemcpyDeviceToHost));

	// Cleanup
	checkCuda(hipEventDestroy(startEvent));
	checkCuda(hipEventDestroy(stopEvent));
	free_call_data(&data);

	return (int) ms;
}

int run_D1_copy_coal12_5() {

	int datasize = BLOCKDIM_X * GRIDDIM_X * 8;


	// Prepare Kernel dimensions
	dim3 dimGrid(GRIDDIM_X, 1, 1);
	dim3 dimBlock(BLOCKDIM_X, 1, 1);

	struct call_data data;
	init_call_data(&data, datasize);

	// Events for timing
	hipEvent_t startEvent, stopEvent;
	checkCuda(hipEventCreate(&startEvent));
	checkCuda(hipEventCreate(&stopEvent));
	float ms;

	// Run Kernel
	checkCuda(hipEventRecord(startEvent, 0));
	D1_copy_coal12_5<<<dimGrid, dimBlock>>>(data.d_odata, data.d_idata);
	checkCuda(hipGetLastError());
	checkCuda(hipEventRecord(stopEvent, 0));
	checkCuda(hipEventSynchronize(stopEvent));
	checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
	checkCuda(hipMemcpy(data.h_odata, data.d_odata, datasize * sizeof(float), hipMemcpyDeviceToHost));

	// Cleanup
	checkCuda(hipEventDestroy(startEvent));
	checkCuda(hipEventDestroy(stopEvent));
	free_call_data(&data);

	return (int) ms;
}

int run_D1_avg2_coal100() {

	int datasize = BLOCKDIM_X * GRIDDIM_X * 2;

	// Prepare Kernel dimensions
	dim3 dimGrid(GRIDDIM_X, 1, 1);
	dim3 dimBlock(BLOCKDIM_X, 1, 1);
	// printf("datasize: %d\n", datasize);

	struct call_data data;
	init_call_data(&data, datasize);

	// Events for timing
	hipEvent_t startEvent, stopEvent;
	checkCuda(hipEventCreate(&startEvent));
	checkCuda(hipEventCreate(&stopEvent));
	float ms;

	// Run Kernel
	checkCuda(hipEventRecord(startEvent, 0));
	D1_avg2_coal100<<<dimGrid, dimBlock>>>(data.d_odata, data.d_idata);
	checkCuda(hipGetLastError());
	checkCuda(hipEventRecord(stopEvent, 0));
	checkCuda(hipEventSynchronize(stopEvent));
	checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
	checkCuda(hipMemcpy(data.h_odata, data.d_odata, datasize * sizeof(float), hipMemcpyDeviceToHost));

	// Cleanup
	checkCuda(hipEventDestroy(startEvent));
	checkCuda(hipEventDestroy(stopEvent));
	free_call_data(&data);

	return (int) ms;
}

int run_D1_avg2_coal50() {

	int datasize = BLOCKDIM_X * GRIDDIM_X * 2;

	// Prepare Kernel dimensions
	dim3 dimGrid(GRIDDIM_X, 1, 1);
	dim3 dimBlock(BLOCKDIM_X, 1, 1);

	struct call_data data;
	init_call_data(&data, datasize);

	// Events for timing
	hipEvent_t startEvent, stopEvent;
	checkCuda(hipEventCreate(&startEvent));
	checkCuda(hipEventCreate(&stopEvent));
	float ms;

	// Run Kernel
	checkCuda(hipEventRecord(startEvent, 0));
	D1_avg2_coal50<<<dimGrid, dimBlock>>>(data.d_odata, data.d_idata);
	checkCuda(hipGetLastError());
	checkCuda(hipEventRecord(stopEvent, 0));
	checkCuda(hipEventSynchronize(stopEvent));
	checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
	checkCuda(hipMemcpy(data.h_odata, data.d_odata, datasize * sizeof(float), hipMemcpyDeviceToHost));

	// Cleanup
	checkCuda(hipEventDestroy(startEvent));
	checkCuda(hipEventDestroy(stopEvent));
	free_call_data(&data);

	return (int) ms;
}

int run_D1_avg4_coal25() {

	int datasize = BLOCKDIM_X * GRIDDIM_X * 4;

	// Prepare Kernel dimensions
	dim3 dimGrid(GRIDDIM_X, 1, 1);
	dim3 dimBlock(BLOCKDIM_X, 1, 1);

	struct call_data data;
	init_call_data(&data, datasize);

	// Events for timing
	hipEvent_t startEvent, stopEvent;
	checkCuda(hipEventCreate(&startEvent));
	checkCuda(hipEventCreate(&stopEvent));
	float ms;

	// Run Kernel
	checkCuda(hipEventRecord(startEvent, 0));
	D1_avg4_coal25<<<dimGrid, dimBlock>>>(data.d_odata, data.d_idata);
	checkCuda(hipGetLastError());
	checkCuda(hipEventRecord(stopEvent, 0));
	checkCuda(hipEventSynchronize(stopEvent));
	checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
	checkCuda(hipMemcpy(data.h_odata, data.d_odata, datasize * sizeof(float), hipMemcpyDeviceToHost));

	// Cleanup
	checkCuda(hipEventDestroy(startEvent));
	checkCuda(hipEventDestroy(stopEvent));
	free_call_data(&data);

	return (int) ms;
}

int run_D1_avg8_coal12_5() {

	int datasize = BLOCKDIM_X * GRIDDIM_X * 8;

	// Prepare Kernel dimensions
	dim3 dimGrid(GRIDDIM_X, 1, 1);
	dim3 dimBlock(BLOCKDIM_X, 1, 1);

	struct call_data data;
	init_call_data(&data, datasize);

	// Events for timing
	hipEvent_t startEvent, stopEvent;
	checkCuda(hipEventCreate(&startEvent));
	checkCuda(hipEventCreate(&stopEvent));
	float ms;

	// Run Kernel
	checkCuda(hipEventRecord(startEvent, 0));
	D1_avg8_coal12_5<<<dimGrid, dimBlock>>>(data.d_odata, data.d_idata);
	checkCuda(hipGetLastError());
	checkCuda(hipEventRecord(stopEvent, 0));
	checkCuda(hipEventSynchronize(stopEvent));
	checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
	checkCuda(hipMemcpy(data.h_odata, data.d_odata, datasize * sizeof(float), hipMemcpyDeviceToHost));

	// Cleanup
	checkCuda(hipEventDestroy(startEvent));
	checkCuda(hipEventDestroy(stopEvent));
	free_call_data(&data);

	return (int) ms;
}
