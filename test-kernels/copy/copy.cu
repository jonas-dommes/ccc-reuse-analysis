#include "hip/hip_runtime.h"
/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*    notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*    notice, this list of conditions and the following disclaimer in the
*    documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*    contributors may be used to endorse or promote products derived
*    from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <stdio.h>
#include <assert.h>

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline hipError_t checkCuda(hipError_t result) {
    #if defined(DEBUG) || defined(_DEBUG)
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    #endif
    return result;
}


// Check errors and print GB/s
void postprocess(const float *ref, const float *res, int n, float ms) {
    bool passed = true;
    for (int i = 0; i < n; i++) {
        if (res[i] != ref[i]) {
            printf("%d %f %f\n", i, res[i], ref[i]);
            printf("%25s\n", "*** FAILED ***");
            passed = false;
            break;
        }
    }
    if (passed) {
        printf("%20.2f\n", 2 * n * sizeof(float) * 1e-6 * NUM_REPS / ms );
    }
}

// Kernel to copy 1 dimensional array
__global__ void copy_array(float *odata, const float *idata, int work_per_thread, int stride) {
    int id =
}



int main(int argc, char **argv) {

}
