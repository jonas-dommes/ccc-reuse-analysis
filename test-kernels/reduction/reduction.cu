#include "hip/hip_runtime.h"
/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*	notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*	notice, this list of conditions and the following disclaimer in the
*	documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*	contributors may be used to endorse or promote products derived
*	from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <stdio.h>
#include <assert.h>
#include <time.h>

#include "utility.h"


// Naive reduce (interleaved addressing)
__global__ void reduce1(int *d_data) {

	// calc index
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

	// do reduction
	for (unsigned int s = 1; s < blockDim.x; s *= 2) {
		if (tid % (2 * s) == 0) {
			d_data[i] = d_data[i] + d_data[i + s];
		}
		__syncthreads();
	}

	// write result for this block to global mem
	if (tid == 0) {
		d_data[blockIdx.x] = d_data[i];
	}
}


// strided indexing for non-divergent branching (interleaved addressing) --> Bank conflicts
__global__ void reduce2(int *d_data) {

	// calc index
	unsigned int tid = threadIdx.x;
	unsigned int offset = blockIdx.x * blockDim.x;

	// do reduction
	for (unsigned int s = 1; s < blockDim.x; s *= 2)  {
		int index = 2 * s * tid;
		if (index < blockDim.x) {
			d_data[offset + index] += d_data[offset + index + s];
		}
		__syncthreads();
	}

	// write result for this block to global mem
	if (tid == 0){
		d_data[blockIdx.x] = d_data[offset];
	}
}


// Sequential Addressing
__global__ void reduce3(int *d_data) {

	// calc index
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

	// do reduction
	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
		if (tid < s) {
			d_data[i] += d_data[i + s];
		}
		__syncthreads();
	}

	// write result for this block to global mem
	if (tid == 0) {
		d_data[blockIdx.x] = d_data[i];
	}
}


__device__ void warpReduce(volatile int* d_data, int index) {
	d_data[index] += d_data[index + 32];
	d_data[index] += d_data[index + 16];
	d_data[index] += d_data[index +  8];
	d_data[index] += d_data[index +  4];
	d_data[index] += d_data[index +  2];
	d_data[index] += d_data[index +  1];
}


// Unroll last Warp
__global__ void reduce4(int *d_data) {

	// calc index
	unsigned int tid = threadIdx.x;
	unsigned int offset = blockIdx.x * blockDim.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

	// do reduction
	for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
		if (tid < s) {
			d_data[i] += d_data[i + s];
		}
		__syncthreads();
	}

	if (tid < 32) warpReduce(d_data, i);

	// write result for this block to global mem
	if (tid == 0) {
		d_data[blockIdx.x] = d_data[i];
	}
}


int main(int argc, char **argv) {

	int success = 0;
	unsigned int data_remaining = 0;

	unsigned int data_points = 2097152;
	unsigned int dimBlock = 128;

	// Get args
	if (argc == 3) {
		data_points = atoi(argv[1]);
		dimBlock = atoi(argv[2]);
	} else {
		printf("Using default datasize, argc = %d\n", argc);
	}

	// Prepare host data
	int *h_idata = (int*) calloc(data_points, sizeof(int));
	int h_odata = 0;

	// Initialize and calculate reference
	init_random_int(h_idata, data_points);
	int ref = calc_reference_reduce(h_idata, data_points);

	// Prepare device data structures
	int *d_data;
	checkCuda(hipMalloc(&d_data, data_points * sizeof(int)));
	checkCuda(hipMemcpy(d_data, h_idata, data_points * sizeof(int), hipMemcpyHostToDevice));

	// ********************** reduce1 **********************
	data_remaining = data_points;

	while (data_remaining > dimBlock) { // Never ends if dimblock == 1
		printf("Call reduce1<<<%d, %d>>>\n", data_remaining / dimBlock, dimBlock);
		reduce1<<<data_remaining / dimBlock , dimBlock>>>(d_data);
		checkCuda(hipGetLastError());

		data_remaining = data_remaining / dimBlock;
	}

	// Make last reduce (would be more efficient to do on Host)
	if (data_remaining > 1) {
		printf("Call reduce1<<<%d, %d>>> for last reduce\n", 1, data_remaining);
		reduce1<<<1 , data_remaining>>>(d_data);
		checkCuda(hipGetLastError());
	}

	// Copy result back to host (theoretically only need first entry)
	checkCuda(hipMemcpy(&h_odata, d_data, sizeof(int), hipMemcpyDeviceToHost));
	// checkCuda(hipMemcpy(h_odata, d_data, data_points * sizeof(int), hipMemcpyDeviceToHost));

	// Compare to reference
	if (ref != h_odata) {
		printf("Reference= %d\nResult   = %d\n", ref, h_odata);
		success ++;
	} else {
		printf("## Success for reduce1!\n");
	}

	// Prepare for next Kernel
	checkCuda(hipMemcpy(d_data, h_idata, data_points * sizeof(int), hipMemcpyHostToDevice));
	h_odata = 0;
	// *****************************************************


	// ********************** reduce2 **********************
	data_remaining = data_points;

	while (data_remaining > dimBlock) { // Never ends if dimblock == 1
		printf("Call reduce2<<<%d, %d>>>\n", data_remaining / dimBlock, dimBlock);
		reduce2<<<data_remaining / dimBlock , dimBlock>>>(d_data);
		checkCuda(hipGetLastError());

		data_remaining = data_remaining / dimBlock;
	}

	// Make last reduce (would be more efficient to do on Host)
	if (data_remaining > 1) {
		printf("Call reduce2<<<%d, %d>>> for last reduce\n", 1, data_remaining);
		reduce2<<<1 , data_remaining>>>(d_data);
		checkCuda(hipGetLastError());
	}

	// Copy result back to host (theoretically only need first entry)
	checkCuda(hipMemcpy(&h_odata, d_data, sizeof(int), hipMemcpyDeviceToHost));
	// checkCuda(hipMemcpy(h_odata, d_data, data_points * sizeof(int), hipMemcpyDeviceToHost));

	// Compare to reference
	if (ref != h_odata) {
		printf("Reference= %d\nResult   = %d\n", ref, h_odata);
		success ++;
	} else {
		printf("## Success for reduce2!\n");
	}

	// Prepare for next Kernel
	checkCuda(hipMemcpy(d_data, h_idata, data_points * sizeof(int), hipMemcpyHostToDevice));
	h_odata = 0;
	// *****************************************************


	// ********************** reduce3 **********************
	data_remaining = data_points;

	while (data_remaining > dimBlock) { // Never ends if dimblock == 1
		printf("Call reduce3<<<%d, %d>>>\n", data_remaining / dimBlock, dimBlock);
		reduce3<<<data_remaining / dimBlock , dimBlock>>>(d_data);
		checkCuda(hipGetLastError());

		data_remaining = data_remaining / dimBlock;
	}

	// Make last reduce (would be more efficient to do on Host)
	if (data_remaining > 1) {
		printf("Call reduce3<<<%d, %d>>> for last reduce\n", 1, data_remaining);
		reduce3<<<1 , data_remaining>>>(d_data);
		checkCuda(hipGetLastError());
	}

	// Copy result back to host (theoretically only need first entry)
	checkCuda(hipMemcpy(&h_odata, d_data, sizeof(int), hipMemcpyDeviceToHost));
	// checkCuda(hipMemcpy(h_odata, d_data, data_points * sizeof(int), hipMemcpyDeviceToHost));

	// Compare to reference
	if (ref != h_odata) {
		printf("Reference= %d\nResult   = %d\n", ref, h_odata);
		success ++;
	} else {
		printf("## Success for reduce3!\n");
	}

	// Prepare for next Kernel
	checkCuda(hipMemcpy(d_data, h_idata, data_points * sizeof(int), hipMemcpyHostToDevice));
	h_odata = 0;
	// *****************************************************


	// ********************** reduce4 **********************
	data_remaining = data_points;

	while (data_remaining > dimBlock) { // Never ends if dimblock == 1
		printf("Call reduce4<<<%d, %d>>>\n", data_remaining / dimBlock, dimBlock);
		reduce4<<<data_remaining / dimBlock , dimBlock>>>(d_data);
		checkCuda(hipGetLastError());

		data_remaining = data_remaining / dimBlock;
	}

	// Make last reduce (would be more efficient to do on Host)
	if (data_remaining > 1) {
		printf("Call reduce4<<<%d, %d>>> for last reduce\n", 1, data_remaining);
		reduce4<<<1 , data_remaining>>>(d_data);
		checkCuda(hipGetLastError());
	}

	// Copy result back to host (theoretically only need first entry)
	checkCuda(hipMemcpy(&h_odata, d_data, sizeof(int), hipMemcpyDeviceToHost));
	// checkCuda(hipMemcpy(h_odata, d_data, data_points * sizeof(int), hipMemcpyDeviceToHost));

	// Compare to reference
	if (ref != h_odata) {
		printf("Reference= %d\nResult   = %d\n", ref, h_odata);
		success ++;
	} else {
		printf("## Success for reduce4!\n");
	}

	// Prepare for next Kernel
	checkCuda(hipMemcpy(d_data, h_idata, data_points * sizeof(int), hipMemcpyHostToDevice));
	h_odata = 0;
	// *****************************************************




	// Cleanup
	checkCuda(hipFree(d_data));

	free(h_idata);

	if (success != 0) {
		printf("\nERROR: %d reductions failed!\n", success);
	}
}
